#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/device_vector.h>
#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;


__global__ void iota(float* data) {
    auto i = threadIdx.x + blockIdx.x * blockDim.x;
    data[i] = i;
}

// [rank + 2の倍数] 番目のデータの和
__device__ float reduce_sum(cg::thread_group g, float* temp, float acc) {
    auto lane = g.thread_rank();
    for (auto i = g.size() / 2; i > 0; i /= 2) {
        temp[lane] = acc;
        g.sync(); // 全threadのストアが終わるまで待機
        if (lane < i) {
            acc += temp[lane + i];
        }
        g.sync(); // 全threadのロードが終わるまで待機
    }
    return acc; // 0番目スレッドの返り値が完全な合計になる
}

__device__ float thread_sum(float *input, int n) {
    float sum = 0;
    // 4-dim ベクトル化による高速化
    // https://devblogs.nvidia.com/cuda-pro-tip-increase-performance-with-vectorized-memory-access
    for(auto i = blockIdx.x * blockDim.x + threadIdx.x;
        i < n / 4; 
        i += blockDim.x * gridDim.x) {
        auto in = reinterpret_cast<float4*>(input)[i];
        sum += in.x + in.y + in.z + in.w;
    }
    return sum;
}

__global__ void sum_kernel_block(float* sum, float* input, int n) {
    auto my_sum = thread_sum(input, n);

    extern __shared__ int temp[];
    auto g = this_thread_block();
    auto block_sum = reduce_sum(g, temp, my_sum);

    if (g.thread_rank() == 0) atomicAdd(sum, block_sum);
}

int main() {
    thrust::device_vector<float> data(11);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0); // 0番目デバイスの情報取得
    int max_block_size = prop.maxThreadsPerBlock;
    std::cout << max_block_size << std::endl;

    dim3 block_size(32, 1, 1);
    dim3 grid_size((data.size() + block_size.x - 1) / block_size.x, 1, 1);
    iota<<<block_size, grid_size>>>(thrust::raw_pointer_cast(data.data()));
    for (auto d : data) {
        std::cout << d << std::endl;
    }
}
